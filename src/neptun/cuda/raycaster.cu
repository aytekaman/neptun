#include "hip/hip_runtime.h"
#include "raycaster.cuh";
#include <stdio.h>

Ray *d_rays;
IntersectionData* d_intersectdata;
glm::vec3* d_points;
TetMesh32::Tet32* d_tets;
ConstrainedFace* d_cons_faces;
Face* d_faces;
unsigned int old_size = 0;

__global__
void raycast_kernel(Ray *rays, int rays_size, glm::vec3* d_points, TetMesh32::Tet32* d_tets, ConstrainedFace* d_cons_faces, Face* d_faces, IntersectionData *output)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < rays_size)
    {
        Ray ray = rays[i];
        unsigned int id[4];
        glm::vec2 p[4];

        const float sign = copysignf(1.0f, ray.dir.z);

        const float a = -1.0f / (sign + ray.dir.z);
        const float b = ray.dir.x * ray.dir.y * a;

        const glm::vec3 right(1.0f + sign * ray.dir.x * ray.dir.x * a, sign * b, -sign * ray.dir.x);
        const glm::vec3 up(b, sign + ray.dir.y * ray.dir.y * a, -ray.dir.y);

        for (int j = 0; j < 4; j++)
        {
            id[j] = rays[i].source_tet.v[j];
            const glm::vec3 point = d_points[id[j]] - ray.origin;
            p[j].x = glm::dot(right, point);
            p[j].y = glm::dot(up, point);
        }

        signed short outIdx = -1;

        if (p[2].x * p[1].y <= p[2].y * p[1].x && p[1].x * p[3].y <= p[1].y * p[3].x && p[3].x * p[2].y <= p[3].y * p[2].x)
            outIdx = 0;
        else if (p[2].x * p[3].y <= p[2].y * p[3].x && p[3].x * p[0].y <= p[3].y * p[0].x && p[0].x * p[2].y <= p[0].y * p[2].x)
            outIdx = 1;
        else if (p[0].x * p[3].y <= p[0].y * p[3].x && p[3].x * p[1].y <= p[3].y * p[1].x && p[1].x * p[0].y <= p[1].y * p[0].x)
            outIdx = 2;
        else if (p[0].x * p[1].y <= p[0].y * p[1].x && p[1].x * p[2].y <= p[1].y * p[2].x && p[2].x * p[0].y <= p[2].y * p[0].x)
        {
            outIdx = 3;
            swap(id[0], id[1]);
            swapvec2(p[0], p[1]);
        }
        else
        {
            output[i].hit = false;
            return;
        }

        int index = ray.source_tet.n[outIdx];

        while (index >= 0)
        {
            id[outIdx] = id[3];
            id[3] = d_tets[index].x ^ id[0] ^ id[1] ^ id[2];
            const glm::vec3 newPoint = d_points[id[3]] - ray.origin;

            p[outIdx] = p[3];
            p[3].x = glm::dot(right, newPoint);
            p[3].y = glm::dot(up, newPoint);

            //p[3] = basis.project(newPoint);

            if (p[3].x * p[0].y < p[3].y * p[0].x) // copysignf here?
            {
                if (p[3].x * p[2].y >= p[3].y * p[2].x)
                    outIdx = 1;
                else
                    outIdx = 0;
            }
            else if (p[3].x * p[1].y < p[3].y * p[1].x)
                outIdx = 2;
            else
                outIdx = 0;

            //prev_index = index;

            if (id[outIdx] == d_tets[index].v[0])
                index = d_tets[index].n[0];
            else if (id[outIdx] == d_tets[index].v[1])
                index = d_tets[index].n[1];
            else if (id[outIdx] == d_tets[index].v[2])
                index = d_tets[index].n[2];
            else
                index = d_tets[index].n[3];
        }

        if (index != -1)
        {
            index = (index & 0x7FFFFFFF);
            const Face& face = d_faces[index];//d_cons_faces[index].face;

            const glm::vec3 *v = face.vertices;
            const glm::vec3 *n = face.normals;
            const glm::vec2 *t = face.uvs;

            const glm::vec3 e1 = v[1] - v[0];
            const glm::vec3 e2 = v[2] - v[0];
            const glm::vec3 s = ray.origin - v[0];
            const glm::vec3 q = glm::cross(s, e1);
            const glm::vec3 p = glm::cross(ray.dir, e2);
            const float f = 1.0f / glm::dot(e1, p);
            const glm::vec2 bary(f * glm::dot(s, p), f * glm::dot(ray.dir, q));

            /*output[i].position =  ray.origin + f * glm::dot(e2, q) * ray.dir;//***
            output[i].normal = bary.x * n[1] + bary.y * n[2] + (1 - bary.x - bary.y) * n[0];//***
            output[i].uv = bary.x * t[1] + bary.y * t[2] + (1 - bary.x - bary.y) * t[0]; //***
            output[i].tet_idx = d_cons_faces[index].tet_idx;
            output[i].neighbor_tet_idx = d_cons_faces[index].other_tet_idx;*/

            output[i].hit = true;
        }
        else
            output[i].hit = false;
    }
}

void copy_to_gpu(TetMesh32& tet_mesh)
{
    hipFree(d_points);
    hipMalloc(&d_points, tet_mesh.m_points.size() * sizeof(glm::vec3));
    hipMemcpy(d_points, tet_mesh.m_points.data(), tet_mesh.m_points.size() * sizeof(glm::vec3), hipMemcpyHostToDevice);
    hipError_t error = hipGetLastError();
    printf("CUDA error1: %s\n", hipGetErrorString(error));

    hipFree(d_tets);
    hipMalloc(&d_tets, tet_mesh.m_tets.size() * sizeof(TetMesh32::Tet32));
    hipMemcpy(d_tets, tet_mesh.m_tet32s, tet_mesh.m_tets.size() * sizeof(TetMesh32::Tet32), hipMemcpyHostToDevice);
    error = hipGetLastError();
    printf("CUDA error2: %s\n", hipGetErrorString(error));

    hipFree(d_cons_faces);
    hipMalloc(&d_cons_faces, tet_mesh.m_constrained_faces.size() * sizeof(ConstrainedFace));
    hipMemcpy(d_cons_faces, tet_mesh.m_constrained_faces.data(), tet_mesh.m_constrained_faces.size() * sizeof(ConstrainedFace), hipMemcpyHostToDevice);
    error = hipGetLastError();
    printf("CUDA error3: %s\n", hipGetErrorString(error));

    hipFree(d_faces);
    hipMalloc(&d_faces, tet_mesh.faces.size() * sizeof(Face));
    hipMemcpy(d_faces, tet_mesh.faces.data(), tet_mesh.faces.size() * sizeof(Face), hipMemcpyHostToDevice);
    error = hipGetLastError();
    printf("CUDA error4: %s\n", hipGetErrorString(error));
}

void ray_caster_gpu(Ray* rays, unsigned int rays_size, IntersectionData* output)
{
    // Allocate space for device copy of data
    if (old_size != rays_size)
    {
        hipFree(d_rays);
        hipFree(d_intersectdata);
        hipMalloc(&d_rays, rays_size * sizeof(Ray));
        hipMalloc(&d_intersectdata, rays_size * sizeof(IntersectionData));
        //hipError_t error = hipGetLastError();
        old_size = rays_size;
    }

    // Copy inputs to device
    hipMemcpy(d_rays, rays, rays_size * sizeof(Ray), hipMemcpyHostToDevice);
    hipError_t error = hipGetLastError();
    printf("CUDA error0: %s\n", hipGetErrorString(error));

    // Launch kernel on GPU
    int t = 256;
    raycast_kernel <<< rays_size / t, t >>>(d_rays, rays_size, d_points, d_tets, d_cons_faces, d_faces, d_intersectdata);
    error = hipGetLastError();
    printf("CUDA error1: %s\n", hipGetErrorString(error));

    // Copy result back to host
    hipMemcpy(output, d_intersectdata, rays_size * sizeof(IntersectionData), hipMemcpyDeviceToHost);

    /*hipFree(d_rays);
    hipFree(d_intersectdata);*/
}
