#include "hip/hip_runtime.h"
#include "raycaster.cuh";
#include <stdio.h>

int N = 640 * 480;

__global__
void raycast_kernel(/*Scene &a,*/ Ray *rays, int rays_size, IntersectionData *output)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < rays_size)
    {
        if ((i / 8) % 2)
            output[i].hit = 1;
        else
            output[i].hit = 0;
    }
}

/*void copy_to_gpu(const std::vector<Ray> rays, Ray* d_rays)
{
    hipFree(d_rays);
    hipMalloc(&d_rays, rays.size() * sizeof(Ray));
    hipMemcpy(d_rays, rays.data(), rays.size() * sizeof(Ray), hipMemcpyHostToDevice);
}

void copy_from_gpu(const Ray* d_rays, Ray* rays)
{
    hipMemcpy(rays, d_rays, rays.size() * sizeof(Ray), hipMemcpyDeviceToHost);
}*/

void ray_caster_gpu(Scene& scene, std::vector<Ray> rays, std::vector<IntersectionData>& output)
{
    Ray *d_rays;
    IntersectionData* c = new IntersectionData[rays.size()];

    IntersectionData *d_intersectdata;
    
    // Allocate space for device copy of Ray
    hipMalloc(&d_rays, rays.size() * sizeof(Ray));
    hipMalloc(&d_intersectdata, rays.size() * sizeof(IntersectionData));
    hipError_t error = hipGetLastError();
    //printf("CUDA error0: %s\n", hipGetErrorString(error));

    // Copy inputs to device
    hipMemcpy(d_rays, rays.data(), rays.size() * sizeof(Ray), hipMemcpyHostToDevice);
    error = hipGetLastError();
    printf("CUDA error1: %s\n", hipGetErrorString(error));

    // Launch kernel on GPU
    raycast_kernel<<< rays.size() / 1024, 1024>>> (d_rays, rays.size(), d_intersectdata);
    //hipDeviceSynchronize();
    error = hipGetLastError();
    printf("CUDA error2: %s\n", hipGetErrorString(error));

    // Copy result back to host
    //hipMemcpyToArray(c, 0, 0, d_intersectdata, rays.size() * sizeof(IntersectionData), hipMemcpyDeviceToHost);
    hipMemcpy(c, d_intersectdata, rays.size() * sizeof(IntersectionData), hipMemcpyDeviceToHost);
    /*error = hipGetLastError();
    printf("CUDA error3: %s\n", hipGetErrorString(error));*/

    // Cleanup
    hipFree(d_rays);
    hipFree(d_intersectdata);

    printf("%d\n", (int)c[110].hit);
    printf("%d\n", (int)c[1021].hit);
    output.insert(output.begin(), c, c + rays.size());

    delete[] c;
}
